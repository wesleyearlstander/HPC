// includes, system
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include <assert.h>
#include <hip/hip_runtime.h>


#define N (1 << 22)

// Simple utility function to check for CUDA runtime errors
void checkCUDAError(const char *msg) {
    hipError_t err = hipGetLastError();
    if( hipSuccess != err) {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) );
        exit(EXIT_FAILURE);
    }
}

void multiplyCPU(const float *h_v1, const float *h_v2, float *h_out) {
    for (int i = 0; i < N; i++) {
        h_out[i] = h_v1[i] * h_v2[i];
    }
}

void expensiveFunctionCPU(const float *h_v1, const float *h_v2, float *h_out) {
    for (int i = 0; i < N; i++) {
        float a = h_v1[i], b = h_v2[i];
        h_out[i] = (a * b) * (sqrt(a + b) + sqrt(a) + sqrt(b - a) + sqrt(b));
    }
}

__global__ void multiplyGPU(const float *g_v1, const float *g_v2, float *g_out) {
    unsigned int index = threadIdx.x + blockDim.x * blockIdx.x;
    g_out[index] = g_v1[index] * g_v2[index];
}

__global__ void expensiveFunctionGPU(const float *g_v1, const float *g_v2, float *g_out) {
    unsigned int index = threadIdx.x + blockDim.x * blockIdx.x;
    float a = g_v1[index], b = g_v2[index];
    g_out[index] = (a * b) * (sqrt(a + b) + sqrt(a) + sqrt(b - a) + sqrt(b));

}

///////////////////////////////////////////////////////////////////////////////
// Program main
///////////////////////////////////////////////////////////////////////////////
int main( int argc, char** argv) {
    ///////////////////// LOTS OF INITIALIZATION CODE ////////////////////////
    // pointers to host vectors v1 and v1
    float *h_v1, *h_v2;
    // pointers to host output for multiply and expensive
    float *h_multiply_correct, *h_expensive_correct;
    // pointers to store results from gpu functions on the host
    float *h_multiply_out, *h_expensive_out;

    // pointer for device vectors v1 and v2
    float *d_v1, *d_v2;
    // pointers to device output
    float *d_multiply_out, *d_expensive_out;



    // Number of test iterations to use for timing
    int testIterations = 3;

    // allocate memory for pointers
    h_v1 = (float *)malloc(N * sizeof(float));
    h_v2 = (float *)malloc(N * sizeof(float));
    h_multiply_correct  = (float *)malloc(N * sizeof(float));
    h_expensive_correct = (float *)malloc(N * sizeof(float));
    h_multiply_out  = (float *)malloc(N * sizeof(float));
    h_expensive_out = (float *)malloc(N * sizeof(float));

    // allocate memory for device pointers
    hipMalloc( (void **) &d_v1, N * sizeof(float));
    hipMalloc( (void **) &d_v2, N * sizeof(float));
    hipMalloc( (void **) &d_multiply_out, N *sizeof(float));
    hipMalloc( (void **) &d_expensive_out, N *sizeof(float));

    // Initialize v1 and v2
    for (int i = 0; i < N; i++) {
        h_v1[i] = 1.0 * (i % 10000);
        h_v2[i] = 2 * h_v1[i];
    }
    float multiply_time = 0;
    float expensive_time = 0;
    float time = 0;

    // CPU functions over testIterations
    for (int i = 0; i < testIterations; i++) {
        // zero memory for outputs
        memset(h_multiply_correct,  0, N * sizeof(float));
        memset(h_expensive_correct,  0, N * sizeof(float));
        time = clock();
		// run cpu kernel
        multiplyCPU(h_v1, h_v2, h_multiply_correct);
        multiply_time += (clock()-time)/CLOCKS_PER_SEC*1000;
        // run cpu kernel
        time = clock();
        expensiveFunctionCPU(h_v1, h_v2, h_expensive_correct);
        expensive_time += (clock()-time)/CLOCKS_PER_SEC*1000;
    }
    printf("Multiply serial run time: %fms\n", multiply_time / testIterations);
    printf("Expensive serial run time: %fms\n", expensive_time / testIterations);

    //////////////////////// INSERT CODE IN THIS SECTION /////////////////////
    // GPU functions over testIterations
    multiply_time = 0;
    expensive_time = 0;
    for (int i = 0; i < testIterations; i++) {
        // zero output memory
        memset(h_multiply_out, 0, N * sizeof(float));
        memset(h_expensive_out, 0, N * sizeof(float));
        hipMemset(d_multiply_out, 0, N * sizeof(float));
        hipMemset(d_expensive_out, 0, N * sizeof(float));
        // zero input memory
        hipMemset(d_v1, 0, N * sizeof(float));
        hipMemset(d_v2, 0, N * sizeof(float));

        // transfer data to GPU
        hipMemcpy(d_v1, h_v1, N * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_v2, h_v2, N * sizeof(float), hipMemcpyHostToDevice);

        hipEvent_t multiply_begin, multiply_end, expensive_begin, expensive_end;
        hipEventCreate(&multiply_begin);
        hipEventCreate(&multiply_end);
        hipEventCreate(&expensive_begin);
        hipEventCreate(&expensive_end);

        hipEventRecord(multiply_begin, 0);
        const size_t block_size = 1024;
        size_t grid_size = N / block_size;
        if(N % block_size) ++grid_size;
        multiplyGPU<<<grid_size, block_size>>>(d_v1, d_v2, d_multiply_out);
        hipEventRecord(multiply_end, 0);
        hipEventSynchronize(multiply_end);

	    hipEventElapsedTime(&time, multiply_begin, multiply_end);
        multiply_time += time;

        // transfer data from GPU
        hipMemcpy(h_multiply_out, d_multiply_out, N * sizeof(float), hipMemcpyDeviceToHost);


        // Check for any CUDA errors
        checkCUDAError("multiplyGPU");

        // zero input memory
        hipMemset(d_v1, 0, N * sizeof(float));
        hipMemset(d_v2, 0, N * sizeof(float));

        // transfer data to GPU
        hipMemcpy(d_v1, h_v1, N * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_v2, h_v2, N * sizeof(float), hipMemcpyHostToDevice);

        const size_t block_size2 = 128;
        size_t grid_size2 = N / block_size2;
        if(N % block_size2) ++grid_size2;

        hipEventRecord(expensive_begin, 0);
        expensiveFunctionGPU<<<grid_size2, block_size2>>>(d_v1, d_v2, d_expensive_out);
        hipEventRecord(expensive_end, 0);
        hipEventSynchronize(expensive_end);

        hipEventElapsedTime(&time, expensive_begin, expensive_end);
        expensive_time += time;

        // transfer data from GPU
        hipMemcpy(h_expensive_out, d_expensive_out, N * sizeof(float), hipMemcpyDeviceToHost);

        // Check for any CUDA errors
        checkCUDAError("expensiveFunctionGPU");
    }
    printf("Multiply kernel run time: %fms\n", multiply_time/ testIterations);
    printf("Expensive kernel run time: %fms\n", expensive_time/ testIterations);
    /////////////////////////////// VALIDATION ///////////////////////////////

    // check if output from gpu kernels is correct
    for (int i = 0; i < N; i++) {
        if (!(abs(h_multiply_out[i] - h_multiply_correct[i]) <= 0.0001)) {
            printf("Test failed (h_multiply_out[%d]:%f != h_multiply_correct[%d]:%f)!\n",
                   i, h_multiply_out[i], i, h_multiply_correct[i]);
            exit(1);
        }
        if (!(abs(h_expensive_out[i] - h_expensive_correct[i]) <= 0.00001 * abs(h_expensive_correct[i]))) {
            printf("Test failed (h_expensive_out[%d]:%f != h_expensive_correct[%d]:%f)!\n",
                   i, h_expensive_out[i], i, h_expensive_correct[i]);
            exit(1);
        }
    }

    printf("Test passed!\n");

    //////////////////////////////// CLEANUP /////////////////////////////////
    // free host memory
    free(h_v1);
    free(h_v2);
    free(h_multiply_correct);
    free(h_expensive_correct);
    free(h_multiply_out);
    free(h_expensive_out);

    // free device memory
    hipFree(d_v1);
    hipFree(d_v2);
    hipFree(d_multiply_out);
    hipFree(d_expensive_out);
    return 0;
}
